#include "hip/hip_runtime.h"

__constant__ int legenda1DH[2500];
__constant__ int velikostiLegendH[100];
__constant__ int posunyLegendH[100];

__constant__ int legenda1DL[2500];
__constant__ int velikostiLegendL[100];
__constant__ int posunyLegendL[100];

__constant__ int vyskaSirka[3]; // vyska sirka

__device__ int countFitness(int * tajenkaPopulace);
__device__ int fitnessColumn(int * tajenkaPopulace, int index, int sloupec);
__device__ int fitnessRow(int* tajenkaPopulace, int index, int radek);
__device__ int needleman(int* Legenda1D, int * velikostiLegend, int * posunyLegend, int* tajenkaVyrezKomprimovany,  int sloupRad, int velikostTajenky);

extern "C"
__global__ void countFitness(int * tajenkaPopulace, int * fitness)
{
    int ind = blockDim.x * blockIdx.x + threadIdx.x ;
    int p1 = ind * vyskaSirka[2];
    fitness[ind]  = countFitness(p1);  // TODO parallel^2
}


__device__ int countFitness(int * tajenkaPopulace){

    int fitness = 0;

    for (int i = 0; i < vyskaSirka[1]; i++) { // sloupce
        fitness += fitnessColumn(tajenkaPopulace, 0, i);
    }

    for (int i = 0; i < vyskaSirka[0]; i++) { // radky
        fitness += fitnessRow(tajenkaPopulace, 0, i);
    }

    return fitness;
}


__device__ int fitnessColumn(int * tajenkaPopulace, int index, int sloupec){

    int velikostTajenky = 1;
    int kombo = 0;

    int * tajenkaVyrezKomprimovany = new int [vyskaSirka[0]];
    tajenkaVyrezKomprimovany[0] = 0;

    for (int i = 0; i < vyskaSirka[0]; i++) {
        if (tajenkaPopulace[index + i*vyskaSirka[1] + sloupec] == 1) {
            kombo++;
        } else {
            if (kombo != 0) {
                tajenkaVyrezKomprimovany[velikostTajenky++] = kombo;
            }
            kombo = 0;
        }
    }

    if (kombo != 0) {
        tajenkaVyrezKomprimovany[velikostTajenky++] = kombo; // posledni ctverecek je cerny
    }
    return needleman(legenda1DL,        velikostiLegendL,      posunyLegendL,       tajenkaVyrezKomprimovany,       sloupec,     velikostTajenky);
}

__device__ int fitnessRow(int* tajenkaPopulace, int index, int radek){

    int velikostTajenky = 1;
    int kombo = 0;

    int * tajenkaVyrezKomprimovany = new int [vyskaSirka[1]];
    tajenkaVyrezKomprimovany[0] = 0;

    for (int i = 0; i < vyskaSirka[1]; i++) {
        if (tajenkaPopulace[index + radek*vyskaSirka[0] + i] == 1) {
            kombo++;
        } else {
            if (kombo != 0) {
                tajenkaVyrezKomprimovany[velikostTajenky++] = kombo;
            }
            kombo = 0;
        }
    }

    if (kombo != 0) {
        tajenkaVyrezKomprimovany[velikostTajenky++] = kombo; // posledni ctverecek je cerny
    }

    return needleman(legenda1DH,        velikostiLegendH,      posunyLegendH,       tajenkaVyrezKomprimovany,       radek,      velikostTajenky);
}


__device__ int needleman(int* legenda1D, int * velikostiLegend, int * posunyLegend, int* tajenkaVyrezKomprimovany,  int sloupRad, int velikostTajenky){

    int velikostLegendy = velikostiLegendH[sloupRad];
    int zacatekLegendy = posunyLegendH[sloupRad];

    int ** H = new int*[velikostTajenky];

    for(int i =0; i < velikostTajenky; i++){
        H[i] = new int[velikostLegendy];
    }

    H[0][0] = 0;

    for (int i = 1; i < velikostTajenky; i++) {
        H[i][0] = H[i - 1][0] - tajenkaVyrezKomprimovany[i];
    }

    for (int i = 1; i < velikostLegendy ; i++) {
        H[0][i] = H[0][i - 1] - legenda1D[zacatekLegendy+i];
    }

    //---------------

    for (int j = 1; j < velikostLegendy; j++) {

        int legendaJ = legenda1DH[ zacatekLegendy+ j];

        for (int i = 1; i < velikostTajenky; i++) {

            H[i][j] = max(H[i - 1][j    ] - tajenkaVyrezKomprimovany[i],
                      max(H[i    ][j - 1] - legendaJ,
                          H[i - 1][j - 1] - abs(legendaJ - tajenkaVyrezKomprimovany[i])));
        }
    }

    for(int i =0 ; i < velikostTajenky;i++){
        free(H[i]);
    }

    free(H);
    free(tajenkaVyrezKomprimovany);

    return H[velikostTajenky - 1][velikostLegendy - 1];
}


