
#include <hip/hip_runtime.h>

__constant__ int legenda1DH[2500];
__constant__ int velikostiLegendH[100];
__constant__ int posunyLegendH[100];

__constant__ int legenda1DL[2500];
__constant__ int velikostiLegendL[100];
__constant__ int posunyLegendL[100];

__constant__ int vyskaSirka[3]; // vyska sirka


extern "C"
__global__ void fitnessPerColumn(int * tajenka1D, int * fitness)
{

    int sloupec = blockDim.x * blockIdx.x + threadIdx.x ;

    int velikostTajenky = 1;

    int * tajenkaSloupec = new int [vyskaSirka[0]];

    tajenkaSloupec[0] = 0;
    int kombo = 0;

    for (int i = 0; i < vyskaSirka[0]; i++) {
        if (tajenka1D[i*vyskaSirka[1] + sloupec] == 1) {
            kombo++;
        } else {
            if (kombo != 0) {
                tajenkaSloupec[velikostTajenky++] = kombo;
            }
            kombo = 0;
        }
    }

    if (kombo != 0) {
        tajenkaSloupec[velikostTajenky++] = kombo; // posledni ctverecek je cerny
    }

    //////////////////////////////////////////////////

    int velikostLegendy = velikostiLegendH[sloupec];
    int zacatekLegendy = posunyLegendH[sloupec];

    int ** H = new int*[velikostTajenky];

    for(int i = 0; i < velikostTajenky; i++){
        H[i] = new int[velikostLegendy];
    }

    H[0][0] = 0;

    for (int i = 1; i < velikostTajenky; i++) {
        H[i][0] = H[i - 1][0] - tajenkaSloupec[i];
    }

    for (int i = 1; i < velikostLegendy ; i++) {
        H[0][i] = H[0][i - 1] - legenda1DH[zacatekLegendy+i];
    }

    //---------------

    for (int j = 1; j < velikostLegendy; j++) {
        int legendaJ = legenda1DH[ zacatekLegendy+ j];

        for (int i = 1; i < velikostTajenky; i++) {

            H[i][j] = max(H[i - 1][j    ] - tajenkaSloupec[i],
                      max(H[i    ][j - 1] - legendaJ,
                          H[i - 1][j - 1] - abs(legendaJ - tajenkaSloupec[i])));
        }
    }


    atomicAdd(fitness, H[velikostTajenky - 1][velikostLegendy - 1]);

    for(int i =0 ; i < velikostTajenky;i++){
        free(H[i]);
    }

    free(H);
    free(tajenkaSloupec);
}



__device__ int getfive(){
    return 5;
}


