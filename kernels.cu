
#include <hip/hip_runtime.h>

__constant__ int legendU[2500]; // upper legends, concatenated
__constant__ int sizesOfLegendsU[100]; // sizes of each of upper legends
__constant__ int shiftsOfLegendsU[100]; // prefix sums of sizes, e.g. where legends begins

__constant__ int legendL[2500]; // left legends, concatenated
__constant__ int sizesOfLegendsL[100]; // sizes of each of left legends
__constant__ int shiftsOfLegendsL[100]; // prefix sums of sizes, e.g. where legends begins

__constant__ int heightWidth[4]; // height, width, height*width of puzzle, population size
__constant__ int numberOfMutations = 4;

// optimize
__device__ void mutate(int * gridPopulation, int * randomCross, int index);
__device__ void cross(int * gridPopulation, int * gridChildren, int i1, int i2, int * randomCross);
__device__ int* copy(int * gridPopulation, int i1); // just for testing

// count fintess
__device__ void needlemanParallel(int * fitness, int* legend1D, int  sizeOfLegend, int beginningOfLegend, int * sliceIntRepr, int sliceSize);

// update
__device__ void overwrite( int * gridPopulation, int * gridChildren, int parent, int child);
__device__ int difference(int * population, int * children, int p, int c);

// creates new generation of individuals and mutates them
// gridPopulation - current population
// gridChildren - array, in which new generation will be stored
// randomCross - random permutation of 1.. size of grid, determines which bits will be copied from which parent.
//                  also used for selection which bits to mutate.
// randomSelection - random permutation 1.. number of individuals, determines touples of parents
extern "C"
__global__ void createChildren(int * gridPopulation, int * gridChildren, int * randomCross, int * randomSelection)
{
    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population

    int p1 = ind * heightWidth[2];       // beginning of grid
    int p2 = randomSelection[ind]* heightWidth[2]; // beginning of grid of the other parent

    cross(gridPopulation, gridChildren, p1, p2, randomCross);

    mutate(gridChildren, randomCross, p1);
}

// replaces parents by children according to their fitness
extern "C"
__global__ void updatePopulation(int * gridPopulation, int * gridChildren, int * fitness, int * fitnessChildren, int * randomSelection)
{
    __shared__ int differenceArray[256];

    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population
    int ind2 = randomSelection[ind]; // the other individual

    int p,pp,c,cc;

    // each children computes difference of two pairs of child and parents
    if(ind < ind2 ){ // p1+c2, p2+c1
        p = ind;
        c = ind2;
        pp = ind2;
        cc = ind;
    }else{           // p1+c1, c2+p2
        p = ind;
        c = ind;
        pp = ind2;
        cc = ind2;
    }

    differenceArray[ind] = difference(gridPopulation, gridChildren, p, c);
    differenceArray[ind] += difference(gridPopulation, gridChildren, pp, cc);

    __syncthreads();

    if(differenceArray[ind] > differenceArray[ind2]){ // should I compare with direct or other parent?
         if (fitness[p] <= fitnessChildren[c]) {
          fitness[p] = fitnessChildren[c];
         p = p * heightWidth[2]; // index in population -> index in array
         c = c * heightWidth[2];
         cc = cc * heightWidth[2];
         overwrite(gridPopulation, gridChildren, p, c);

        }
    } else {
       if (fitness[p] <= fitnessChildren[cc]) {
            fitness[p] = fitnessChildren[cc];
            p = p * heightWidth[2]; // index in population -> index in array
            c = c * heightWidth[2];
            cc = cc * heightWidth[2];
           overwrite(gridPopulation, gridChildren, p, cc);

       }
    }
}


// puts together a child from two parents, individuals bits are determined by permutation randomCross
__device__ void cross(int * gridPopulation, int * gridChildren, int parent1, int parent2, int * randomCross){

    for (int i = 0; i < heightWidth[2]/2 ; i++) {
        gridChildren[parent1 + randomCross[i]] = gridPopulation[parent1 + randomCross[i]];
    }

    for (int i = heightWidth[2]/2; i < heightWidth[2] ; i++) {
        gridChildren[parent1 + randomCross[i]] = gridPopulation[parent2 + randomCross[i]];
    }
}

// changes few bits in new individual
__device__ void mutate(int * gridChildren, int * randomPerm, int index){

    int x = randomPerm[0];

     for (int i = 0; i < numberOfMutations; i++) {

        x = randomPerm[x];
        gridChildren[index + x] = 1 - gridChildren[index + x];
    }
}


// overwrites
__device__ void overwrite( int * gridPopulation, int* gridChildren, int p, int c){
    for (int i = 0; i < heightWidth[2]; i++) {
        gridPopulation[p + i] = gridChildren[c + i];
    }
}

// counts how simmillar is parent to child
__device__ int difference(int * population, int* children, int p, int c){

    int diff = 0;
    p = p*heightWidth[2];
    c = c*heightWidth[2];

    for (int i = 0; i < heightWidth[2]; i++) {
        if (children[c + i] != population[p + i]) {
            diff++;
        }
    }

    return diff;
}

// creates representation of column, which is comparable to legend and runs NW function
extern "C"
__global__ void countFitnessOfAllColumns(int * gridPopulation, int * fitness){

    int columnIndex = blockDim.x * blockIdx.x + threadIdx.x ; // column number relative to all grids in population
    int numberOfIndividual = columnIndex / heightWidth[1]; // number in population
    int absStart = numberOfIndividual * heightWidth[2];  // start in array of population
    int column = columnIndex - numberOfIndividual * heightWidth[1]; // relative to grid

    int sliceSize = 1;

    int * columnIntRepr = new int [heightWidth[0]];

    columnIntRepr[0] = 0;
    int combo = 0;

    for (int i = 0; i < heightWidth[0]; i++) {
        if (gridPopulation[absStart + i*heightWidth[1] + column] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                columnIntRepr[sliceSize++] = combo;
            }
            combo = 0;
        }
    }
    if (combo != 0) {
        columnIntRepr[sliceSize++] = combo; // for the case the last square is filled
    }

    needlemanParallel(&fitness[numberOfIndividual],legendU, sizesOfLegendsU[column], shiftsOfLegendsU[column], columnIntRepr, sliceSize);
}


// creates representation of row, which is comparable to legend and runs NW function
extern "C"
__global__ void countFitnessOfAllRows(int * gridPopulation, int * fitness){

    int absRadek = blockDim.x * blockIdx.x + threadIdx.x ;
    int numberOfIndividual = absRadek / heightWidth[0];
    int absStart = numberOfIndividual * heightWidth[2];  // index v tajence populace
    int row = absRadek - numberOfIndividual * heightWidth[0];

    int sliceSize = 1;
    int combo = 0;

    int * rowIntRepr = new int [heightWidth[1]];
    rowIntRepr[0] = 0;

    for (int i = 0; i < heightWidth[1]; i++) {
        if (gridPopulation[absStart + row*heightWidth[1] + i] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                rowIntRepr[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        rowIntRepr[sliceSize++] = combo; // for the case the last square is filled
    }

    needlemanParallel(&fitness[numberOfIndividual], legendL, sizesOfLegendsL[row], shiftsOfLegendsL[row], rowIntRepr, sliceSize);
}

// computes Needleman-Wunsch function, which measures the difference between two integer arrays and adds it to fitness.
// here, one array is legend of one row/column and the other is actual slice of individual's grid.
__device__ void needlemanParallel(int * fitness, int* legend1D, int  sizeOfLegend, int beginningOfLegend, int* sliceIntRepr, int sliceSize){

    int ** H = new int*[sliceSize];

    for(int i = 0; i < sliceSize; i++){
        H[i] = new int[sizeOfLegend];
    }

    H[0][0] = 0;

    for (int i = 1; i < sliceSize; i++) {
        H[i][0] = H[i - 1][0] - sliceIntRepr[i];
    }

    for (int i = 1; i < sizeOfLegend ; i++) {
        H[0][i] = H[0][i - 1] - legend1D[beginningOfLegend+i];
    }

    //---------------

    for (int j = 1; j < sizeOfLegend; j++) {

        int legendJ = legend1D[ beginningOfLegend+ j];

        for (int i = 1; i < sliceSize; i++) {

            H[i][j] = max(H[i - 1][j    ] - sliceIntRepr[i],
                      max(H[i    ][j - 1] - legendJ,
                          H[i - 1][j - 1] - abs(legendJ - sliceIntRepr[i])));
        }
    }

    atomicAdd(fitness, H[sliceSize - 1][sizeOfLegend - 1]);

    for(int i =0 ; i < sliceSize;i++){
        free(H[i]);
    }

    free(H);
    free(sliceIntRepr);
}






///
/// Single thread computation of fitness

__device__ int countFitness(int * gridPopulation, int index);
__device__ int fitnessColumn(int * gridPopulation, int index, int column);
__device__ int fitnessRow(int * gridPopulation, int index, int row);
__device__ int needleman(int* legend1D, int  sizeOfLegend, int beginningOfLegend, int* gridSlice, int sliceSize);

extern "C"
__global__ void evolution(int * gridPopulation, int * gridChildren, int * fitness, int * fitnessChildren, int* randomCross, int * randomSelection){

     __shared__ int differenceArray[256];

    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population
    int ind2 = randomSelection[ind]; // the other individual



   cross(gridPopulation, gridChildren, ind * heightWidth[2], ind2 * heightWidth[2], randomCross);

   mutate(gridChildren, randomCross, ind * heightWidth[2]);



    fitnessChildren[ind] = countFitness(gridChildren, ind * heightWidth[2]);


    __syncthreads();

    int p,pp,c,cc;

    // each children computes difference of two pairs of child and parents
    if(ind < ind2 ){ // p1+c2, p2+c1
        p = ind;
        c = ind2;
        pp = ind2;
        cc = ind;
    }else{           // p1+c1, c2+p2
        p = ind;
        c = ind;
        pp = ind2;
        cc = ind2;
    }

    differenceArray[ind] = difference(gridPopulation, gridChildren, p, c);
    differenceArray[ind] += difference(gridPopulation, gridChildren, pp, cc);

    __syncthreads();

    if(differenceArray[ind] > differenceArray[ind2]){ // should I compare with direct or other parent?

         if (fitness[p] <= fitnessChildren[c]) {
             fitness[p] = fitnessChildren[c];
             overwrite(gridPopulation, gridChildren, ind * heightWidth[2], c* heightWidth[2]);
        }

    } else {
       if (fitness[p] <= fitnessChildren[cc]) {
            fitness[p] = fitnessChildren[cc];
           overwrite(gridPopulation, gridChildren, ind * heightWidth[2], cc* heightWidth[2]);
       }
    }



}

__device__ int countFitness(int * gridPopulation, int index ){

    int fitness = 0;

    for (int column = 0; column < heightWidth[1]; column++) { // sloupce
        fitness += fitnessColumn(gridPopulation, index, column);
    }

    for (int row = 0; row < heightWidth[0]; row++) { // radky
        fitness += fitnessRow(gridPopulation, index, row);
    }

    return fitness;
}


__device__ int fitnessColumn(int * gridPopulation, int index, int column){

    int sliceSize = 1;
    int combo = 0;

    int * gridSlice = new int [heightWidth[0]];
    gridSlice[0] = 0;

    for (int i = 0; i < heightWidth[0]; i++) {
        if (gridPopulation[index + i*heightWidth[1] + column] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                gridSlice[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        gridSlice[sliceSize++] = combo; // for the case the last square is filled
    }

    return needleman(legendU,        sizesOfLegendsU[column], shiftsOfLegendsU[column],  gridSlice,    sliceSize);
}


__device__ int fitnessRow(int * gridPopulation, int index, int row){

    int sliceSize = 1;
    int combo = 0;

    int * gridSlice = new int [heightWidth[1]];
    gridSlice[0] = 0;

    for (int i = 0; i < heightWidth[1]; i++) {
        if (gridPopulation[index + row*heightWidth[1] + i] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                gridSlice[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        gridSlice[sliceSize++] = combo; // for the case the last square is filled
    }

    return needleman(legendL, sizesOfLegendsL[row],  shiftsOfLegendsL[row],  gridSlice,   sliceSize);
}


__device__ int needleman(int* legend1D, int sizeOfLegend, int beginningOfLegend, int* gridSlice, int sliceSize){

    int ** H = new int*[sliceSize];

    for(int i =0; i < sliceSize; i++){
        H[i] = new int[sizeOfLegend];
    }

    H[0][0] = 0;

    for (int i = 1; i < sliceSize; i++) {
        H[i][0] = H[i - 1][0] - gridSlice[i];
    }

    for (int i = 1; i < sizeOfLegend ; i++) {
        H[0][i] = H[0][i - 1] - legend1D[beginningOfLegend+i];
    }

    //---------------

    for (int j = 1; j < sizeOfLegend; j++) {

        int legendJ = legend1D[ beginningOfLegend+ j];

        for (int i = 1; i < sliceSize; i++) {

            H[i][j] = max(H[i - 1][j    ] - gridSlice[i],
                      max(H[i    ][j - 1] - legendJ,
                          H[i - 1][j - 1] - abs(legendJ - gridSlice[i])));
        }
    }

    int subFitness = H[sliceSize - 1][sizeOfLegend - 1];

    for(int i =0 ; i < sliceSize;i++){
        free(H[i]);
    }

    free(H);
    free(gridSlice);

    return subFitness;

}
