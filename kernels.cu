
#include <hip/hip_runtime.h>

// Constant memory
__constant__ int legendU[2500]; // upper legends, concatenated
__constant__ int sizesOfLegendsU[100]; // sizes of each of upper legends
__constant__ int shiftsOfLegendsU[100]; // prefix sums of sizes, e.g. where legends begins

__constant__ int legendL[2500]; // left legends, concatenated
__constant__ int sizesOfLegendsL[100]; // sizes of each of left legends
__constant__ int shiftsOfLegendsL[100]; // prefix sums of sizes, e.g. where legends begins

__constant__ int heightWidth[4]; // height, width, height*width of puzzle, population size
__constant__ int numberOfMutations = 4;

// create offspring
__device__ void mutate(int * gridPopulation, int * randomCross, int index);
__device__ void cross(int * gridPopulation, int * gridChildren, int i1, int i2, int * randomCross);

// count fintess
__device__ void needlemanParallel(int * fitness, int* legend1D, int  sizeOfLegend, int beginningOfLegend, int * gridSlice, int sliceSize);

/// Single thread computation of fitness
__device__ int countFitness(int * gridPopulation, int index);
__device__ int fitnessColumn(int * gridPopulation, int index, int column, int * H0, int * H1, int * gridSlice);
__device__ int fitnessRow(int * gridPopulation, int index, int row, int * H0, int * H1, int * gridSlice);
__device__ int needlemanOpt(int* legend1D, int  sizeOfLegend, int beginningOfLegend, int* gridSlice, int sliceSize, int * H0, int * H1);


// creates new generation of individuals and mutates them
// gridPopulation - current population
// gridChildren - array, in which new generation will be stored
// randomCross - random permutation of 1.. size of grid, determines which bits will be copied from which parent.
//                  also used for selection which bits to mutate.
// randomSelection - random permutation 1.. number of individuals, determines touples of parents
extern "C"
__global__ void createChildren(int * gridPopulation, int * gridChildren, int * randomCross, int * randomSelection)
{
    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population

    int p1 = ind * heightWidth[2];       // beginning of grid
    int p2 = randomSelection[ind]* heightWidth[2]; // beginning of grid of the other parent

    cross(gridPopulation, gridChildren, p1, p2, randomCross);

    mutate(gridChildren, randomCross, p1);
}


// puts together a child from two parents, individuals bits are determined by permutation randomCross
__device__ void cross(int * gridPopulation, int * gridChildren, int parent1, int parent2, int * randomCross){

    for (int i = 0; i < heightWidth[2]/2 ; i++) {
        gridChildren[parent1 + randomCross[i]] = gridPopulation[parent1 + randomCross[i]];
    }

    for (int i = heightWidth[2]/2; i < heightWidth[2] ; i++) {
        gridChildren[parent1 + randomCross[i]] = gridPopulation[parent2 + randomCross[i]];
    }
}

// changes few bits in new individual
__device__ void mutate(int * gridChildren, int * randomPerm, int index){

    int x = randomPerm[0];

     for (int i = 0; i < numberOfMutations; i++) {

        x = randomPerm[x];
        gridChildren[index + x] = 1 - gridChildren[index + x];
    }
}


// creates representation of column, which is comparable to legend and runs NW function
extern "C"
__global__ void countFitnessOfAllColumns(int * gridPopulation, int * fitness){

    int columnIndex = blockDim.x * blockIdx.x + threadIdx.x ; // column number relative to all grids in population
    int numberOfIndividual = columnIndex / heightWidth[1]; // number in population
    int absStart = numberOfIndividual * heightWidth[2];  // start in array of population
    int column = columnIndex - numberOfIndividual * heightWidth[1]; // relative to grid

    int sliceSize = 1;

    int * columnIntRepr = new int [heightWidth[0]];

    columnIntRepr[0] = 0;
    int combo = 0;

    for (int i = 0; i < heightWidth[0]; i++) {
        if (gridPopulation[absStart + i*heightWidth[1] + column] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                columnIntRepr[sliceSize++] = combo;
            }
            combo = 0;
        }
    }
    if (combo != 0) {
        columnIntRepr[sliceSize++] = combo; // for the case the last square is filled
    }

    needlemanParallel(&fitness[numberOfIndividual],legendU, sizesOfLegendsU[column], shiftsOfLegendsU[column], columnIntRepr, sliceSize);
}


// creates representation of row, which is comparable to legend and runs NW function
extern "C"
__global__ void countFitnessOfAllRows(int * gridPopulation, int * fitness){

    int absRadek = blockDim.x * blockIdx.x + threadIdx.x ;
    int numberOfIndividual = absRadek / heightWidth[0];
    int absStart = numberOfIndividual * heightWidth[2];  // index v tajence populace
    int row = absRadek - numberOfIndividual * heightWidth[0];

    int sliceSize = 1;
    int combo = 0;

    int * rowIntRepr = new int [heightWidth[1]];
    rowIntRepr[0] = 0;

    for (int i = 0; i < heightWidth[1]; i++) {
        if (gridPopulation[absStart + row*heightWidth[1] + i] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                rowIntRepr[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        rowIntRepr[sliceSize++] = combo; // for the case the last square is filled
    }

    needlemanParallel(&fitness[numberOfIndividual], legendL, sizesOfLegendsL[row], shiftsOfLegendsL[row], rowIntRepr, sliceSize);
}

// computes Needleman-Wunsch function, which measures the difference between two integer arrays and adds it to fitness.
// here, one array is legend of one row/column and the other is actual slice of individual's grid.
__device__ void needlemanParallel(int * fitness, int* legend, int sizeOfLegend, int shiftsOfLegends, int* sliceIntRepr, int sliceSize){

    int * H0 = new int[sliceSize];
    int * H1 = new int[sliceSize];

    int fitnessLocal = needlemanOpt(legend, sizeOfLegend,  shiftsOfLegends,  sliceIntRepr, sliceSize, H0, H1);

    atomicAdd(fitness, fitnessLocal);

    free(sliceIntRepr);
    free(H0);
    free(H1);
}




///
/// Single thread computation of evolution

extern "C"
__global__ void evolution(int * gridPopulation, int * gridChildren, int * fitness, int* fitnessChildren, int* randomCross, int * randomSelection){

    __shared__ int differenceArray[320];

    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population
    int ind2 = randomSelection[ind]; // the other individual

    cross(gridPopulation, gridChildren, ind * heightWidth[2], ind2 * heightWidth[2], randomCross);

    mutate(gridChildren, randomCross, ind * heightWidth[2]);

    fitnessChildren[ind] = countFitness(gridChildren, ind * heightWidth[2]);

}

__device__ int countFitness(int * gridPopulation, int index ){

    int fitness = 0;
    int biggerSize = max(heightWidth[0], heightWidth[1])/2; // take the maximal possible size

    int * H0 = new int[biggerSize];
    int * H1 = new int[biggerSize];

    int * gridSlice = new int [biggerSize];
    gridSlice[0] = 0;

    for (int column = 0; column < heightWidth[1]; column++) { // sloupce
        fitness += fitnessColumn(gridPopulation, index, column, H0, H1, gridSlice);
    }

    for (int row = 0; row < heightWidth[0]; row++) { // radky
        fitness += fitnessRow(gridPopulation, index, row, H0, H1, gridSlice);
    }

    free(gridSlice);
    free(H0);
    free(H1);

    return fitness;
}


__device__ int fitnessColumn(int * gridPopulation, int index, int column, int * H0, int * H1, int * gridSlice){

    int sliceSize = 1;
    int combo = 0;

    gridSlice[0] = 0;

    for (int i = 0; i < heightWidth[0]; i++) {
        if (gridPopulation[index + i*heightWidth[1] + column] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                gridSlice[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        gridSlice[sliceSize++] = combo; // for the case the last square is filled
    }

    return needlemanOpt(legendU,        sizesOfLegendsU[column], shiftsOfLegendsU[column],  gridSlice,    sliceSize,  H0,  H1);
}


__device__ int fitnessRow(int * gridPopulation, int index, int row, int * H0, int * H1, int * gridSlice){

    int sliceSize = 1;
    int combo = 0;

    for (int i = 0; i < heightWidth[1]; i++) {
        if (gridPopulation[index + row*heightWidth[1] + i] == 1) {
            combo++;
        } else {
            if (combo != 0) {
                gridSlice[sliceSize++] = combo;
            }
            combo = 0;
        }
    }

    if (combo != 0) {
        gridSlice[sliceSize++] = combo; // for the case the last square is filled
    }
    return needlemanOpt(legendL, sizesOfLegendsL[row],  shiftsOfLegendsL[row],  gridSlice,   sliceSize, H0, H1);
}


__device__ int needlemanOpt(int* legend1D, int sizeOfLegend, int beginningOfLegend, int* gridSlice, int sliceSize, int* H0, int *H1){

    H0[0] = 0;
    H1[0] = 0;

    for (int i = 1; i < sliceSize; i++) {
        H0[i] = H0[i - 1] - gridSlice[i];
    }

    //---------------

    for (int j = 1; j < sizeOfLegend; j++) {

        int legendJ = legend1D[beginningOfLegend + j];

        H1[0] = H0[0] - legendJ;

        for (int i = 1; i < sliceSize; i++) {
            H1[i] = max(H1[i-1] - gridSlice[i],
                    max(H0[i  ] - legendJ,
                        H0[i-1] - abs(legendJ - gridSlice[i])));
        }

        int * swap = H0;
        H0 = H1;
        H1 = swap;
    }

    return  H0[sliceSize - 1]; // swapped, so H0;
}