
#include <hip/hip_runtime.h>

__constant__ int heightWidth[4]; // height, width, height*width of puzzle, population size


__device__ void overwrite( int * gridPopulation, int * gridChildren, int parent, int child);
__device__ int difference(int * population, int * children, int p, int c);

// replaces parents by children according to their fitness
extern "C"
__global__ void updatePopulation(int * gridPopulation, int * gridChildren, int * fitness, int * fitnessChildren, int * randomSelection)
{
    __shared__ int differenceArray[320];

    int ind = blockDim.x * blockIdx.x + threadIdx.x ; // number of individual in population
    int ind2 = randomSelection[ind]; // the other individual

    int p,pp,c,cc;

    // each children computes difference of two pairs of child and parents
    if(ind < ind2 ){ // p1+c2, p2+c1
        p = ind;
        c = ind2;
        pp = ind2;
        cc = ind;
    }else{           // p1+c1, c2+p2
        p = ind;
        c = ind;
        pp = ind2;
        cc = ind2;
    }

    differenceArray[ind] = difference(gridPopulation, gridChildren, p, c);
    differenceArray[ind] += difference(gridPopulation, gridChildren, pp, cc);

    __syncthreads();

    if(differenceArray[ind] > differenceArray[ind2]){ // should I compare with direct or other parent?
         if (fitness[p] <= fitnessChildren[c]) {
          fitness[p] = fitnessChildren[c];
         p = p * heightWidth[2]; // index in population -> index in array
         c = c * heightWidth[2];
         cc = cc * heightWidth[2];
         overwrite(gridPopulation, gridChildren, p, c);

        }
    } else {
       if (fitness[p] <= fitnessChildren[cc]) {
            fitness[p] = fitnessChildren[cc];
            p = p * heightWidth[2]; // index in population -> index in array
            c = c * heightWidth[2];
            cc = cc * heightWidth[2];
           overwrite(gridPopulation, gridChildren, p, cc);

       }
    }
}



// overwrites
__device__ void overwrite( int * gridPopulation, int* gridChildren, int p, int c){
    for (int i = 0; i < heightWidth[2]; i++) {
        gridPopulation[p + i] = gridChildren[c + i];
    }
}



// counts how simmillar is parent to child
__device__ int difference(int * population, int* children, int p, int c){

    int diff = 0;
    p = p*heightWidth[2];
    c = c*heightWidth[2];

    for (int i = 0; i < heightWidth[2]; i++) {
        if (children[c + i] != population[p + i]) {
            diff++;
        }
    }

    return diff;
}